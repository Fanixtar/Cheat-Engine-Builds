#include "hip/hip_runtime.h"
/*
Pointerscan core
While the original pointerscan can run forever using recursive loops for each thread, the cuda implementation can not do that.
If a kernel does not exit within 2 seconds, it will crash. So, in order to deal with this, the recusrsive loop hase been replaced with a recursion replacement which
allows resume capability
*/
#include "hip/hip_runtime.h"

#include <windows.h>
#include <stdio.h>

#include "pscanfileaccess.h"
#include "cudapointervaluelist.cuh"

#define MAXCOMMANDLISTSIZE	2048
#pragma pack(16)

typedef __declspec(align(16)) struct _rcaller //recursion replacement
{
	UINT_PTR valueToFind;
	UINT_PTR startvalue;  
	UINT_PTR stopvalue; 
	PPointerList plist;
	int plistIndex;  //index in the plist to start off with
} rcaller, *prcaller;

typedef __declspec(align(16)) struct _workcommand  //same as continuedata but no plist data
{
	UINT_PTR valueToFind;  
	int level;  
	int *offsets; 
} WorkCommand, *PWorkCommand;


__global__ void pscan(PWorkCommand queueElements, PWorkCommand staticoutputqueue, int staticoutputquesize, PWorkCommand *allocatedoutputqueue, int *allocatedoutputsize)
/*
The pointerscanner iteration
*/
{
	int index = threadIdx.x;// 0; // blockIdx.x * blockDim.x + threadIdx.x;
	UINT_PTR stopValue = queueElements[index].valueToFind + 4096;
	PPointerList pl;
	int i;

	pl = findPointerValue(queueElements[index].valueToFind, &stopValue);

	if (pl == NULL)
	{
		staticoutputqueue[index].level = 666;
		staticoutputqueue[index].offsets = 0;
	}
	else
	{
		staticoutputqueue[index].level = 777;
		staticoutputqueue[index].offsets = 0;
	}
}



int pointerscan(UINT_PTR address, int structsize, int maxlevel)
{
	PWorkCommand wc = (PWorkCommand)malloc(sizeof(WorkCommand) * 1024);
	int wcsize = 1;

  int i=0;
  BOOL done=FALSE;  
  hipError_t err;

  //loop till all are done
  /*
  
  while (!done)
  {	  
	  if (i%10==0)
	  {
	    int r=0;	  
		err=hipMemcpyToSymbol(HIP_SYMBOL(didWork), &r, sizeof(r));	
		if (err!=hipSuccess)  
		{
		  printf("CUDA error: %s\n", hipGetErrorString(err));  
	 	  break;
		}
		
		r=12;
		err=hipMemcpyFromSymbol(&r, HIP_SYMBOL(didWork), sizeof(r));
		if (err!=hipSuccess)  
		{
		  printf("CUDA error: %s\n", hipGetErrorString(err));  
	 	  break;
		}
		
		if (r!=0)
		{
			printf("FAIL\n");
			break;
		}		
	  }
	  
	//  printf("------------SCAN %d------------------\n", i);  
	  pscan<<<1,1024>>>(cd, structsize, 5); 
	  hipDeviceSynchronize(); 
	  
	  err=hipGetLastError();
	
	
	
	  if (err!=hipSuccess)  
	  {
		printf("CUDA error: %s\n", hipGetErrorString(err));  
		break;
	  }
	  
	  if (i%10==0)
	  {
	    int r=0;	  
		err=hipMemcpyFromSymbol(&r, HIP_SYMBOL(didWork), sizeof(r));	
		if (err!=hipSuccess)  
		{
		  printf("CUDA error: %s\n", hipGetErrorString(err));  
	 	  break;
		}
				
		if (r==0)
		  done=TRUE;
	  }	  
	  
	  i++;
	  

	  


  }
  
  //loop
  

  */
   
    
  
  
  return 0;
}